
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string>

#include <iostream>

using namespace std;

void delay(int number_of_seconds)
{
    // Converting time into milli_seconds
    int milli_seconds = 1000 * number_of_seconds;
 
    // Storing start time
    clock_t start_time = clock();
 
    // looping till required time is not achieved
    while (clock() < start_time + milli_seconds)
        ;
}

__global__ void square(float * d_out, float * d_in) {
    int idx = threadIdx.x; // threadIdx is a cuda built-in variable
    float f = d_in[idx];
    d_out[idx] = f * f;
}

int main(int argc, char* argv[]) {

	cout << "comes" << endl;
    const int NUM_OF_BLOCKS = stoi(argv[1]);
    const int ARRAY_SIZE = stoi(argv[2]);
    const int ARRAY_BYTES = ((NUM_OF_BLOCKS * ARRAY_SIZE) *  (sizeof(float)));
    float h_in[ARRAY_BYTES];
    
    int counter = 0;

    for (int i = 0; i < ARRAY_SIZE; i++) {
        h_in[i] = float(i);
    }
    float h_out[ARRAY_BYTES];

    float * d_in;
    float * d_out;


    cout << "come here too before allocation" << endl;

    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);
cout << "after allocation and before copy" << endl;

    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	
    
    
    while(counter < 2000000) {
    	square<<<NUM_OF_BLOCKS, ARRAY_SIZE>>>(d_out, d_in);
    	//delay(10);

	counter++;
    }
cout << "before copy back" << endl;
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
    
    return 0;
}
